#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"

#include "KohonenMap.h"
#include "Normalizer.h"

#include <cfloat>
#include <vector>
#include <string>
#include <iostream>
#include <time.h>
#include <ctime>
#include <fstream>
#include <random>
#include <algorithm>
#include <functional>
#include <numeric>
#include <exception>
#include <new>
#include <sstream>
#include  <cctype>

//#define DEBUG

using namespace std;
using namespace std::placeholders;

KohonenMap::KohonenMap(int _numPoints, bool _countHonkong) : numPoints(_numPoints), isTeached(false)
{
	data = new vector<vector<double>>;
	weights = new vector<vector<double>>;
	Normalizer A("billionaires.csv");

	if (_countHonkong)
	{
		A.convertHonkongToChina();

	}
	A.normalize(data);
	size = data->size();
	featureSize = (*data)[0].size();
	howMuchInOneCell = size / numPoints; //������� ������� �������������� � ����� ������
	namesData = { A.getNames() };

	random_device rd;
	mt19937 g(rd());

	vector<size_t> index(size);
	iota(index.begin(), index.end(), 0); //������ ������ ��������, ������� � 0
	shuffle(index.begin(), index.end(), g); //������ ��� ���������

	vector<vector<double>>* dataSafe = new vector<vector<double>>(size, vector<double>(featureSize)); //��������� ������������� �� ������ � ���� �� ���������
	vector<string> namesSafe = vector<string>(size);
	for_each(index.begin(), index.end(), [&](auto& _n)
		{
			size_t i = distance(index.begin(), find(index.begin(), index.end(), _n));

			(*dataSafe)[i] = (*data)[_n]; //��� �����, ����������, �������������� �� ������ ������
			namesSafe[i] = namesData[_n];
		});

	copy(dataSafe->begin(), dataSafe->end(), data->begin()); //������ ������� ���� � ���
	copy(namesSafe.begin(), namesSafe.end(), namesData.begin());


	vector<double> weightsRow(featureSize);
	random_device rd2;
	mt19937 gen(rd2());



	for (size_t i = 0; i < _numPoints; i++) //����, ��� ������������ ���� ����
	{
		vector<double> weightsRow(featureSize);
		weights->push_back(weightsRow);
		clusterization[i + 1] = 0;

		for (size_t j = 0; j < featureSize; j++)
		{
			double maxFeat = findMaxOrMin(j, true);
			double minFeat = findMaxOrMin(j, false);
			if (maxFeat > 1e-7)
			{
				//(*weights)[i][j] = (double)(rand() % ((int)round(maxFeat))) + minFeat;
				if ((minFeat == 0.0) && (maxFeat == 1.0))
				{
					uniform_int_distribution<> dis((int)minFeat, (int)maxFeat);
					(*weights)[i][j] = (double)dis(gen);
				}
				else
				{
					uniform_real_distribution<> dis(minFeat, maxFeat);
					(*weights)[i][j] = dis(gen);
				}
			}
			else
			{
				(*weights)[i][j] = 0.0;
			}
			if (maxFeat != 0)
			{
				(*weights)[i][j] /= maxFeat;
			}


		}
	}

	//distanceSquared.push_back(0.0);
	fill_n(back_inserter(distanceSquared), numPoints, 0.0);

	/*for (size_t i = 0; i < numPoints; i++)
	{
		distanceSquared.push_back(0);
	}*/
	return;

}

vector<vector<double>> KohonenMap::littleChange(size_t whatSamplesToChange, double percent)
{

	vector<vector<double>> result(100, vector<double>(featureSize));
	for_each(data->begin() + (whatSamplesToChange - 1) * 100, data->begin() + (whatSamplesToChange) * 100, [&](auto& _n)
		{
			size_t i = distance(data->begin() + (whatSamplesToChange - 1) * 100,
			find(data->begin() + (whatSamplesToChange - 1) * 100, data->begin() + (whatSamplesToChange) * 100, _n));

	for_each(_n.begin(), _n.end(), [&](auto& _m)
		{
			size_t j = distance(_n.begin(), find(_n.begin(), _n.end(), _m));

			result[i][j] = _m + pow(-1, rand() % 2) * percent * _m / 100;//������� �������� ��������� ������ � ���� ��� ������ �������
			return;
		});
	return;
		});

	/*for (size_t i = (whatSamplesToChange - 1) * 100; whatSamplesToChange * 100 + 1; i++)
	{
		for (size_t j = 0; j < featureSize; j++)
		{
			(*data)[i][j] += pow(-1, rand() % 2) * (*data)[i][j] / 100; //������� �������� ��������� ������ � ���� ��� ������ �������
		}
	}*/
	return result;
}
double const KohonenMap::findMaxOrMin(size_t numFeature, bool isMaxOrMin)
{
	vector<double> arrayFeature(size);
	size_t i = 0;
	for_each(arrayFeature.begin(), arrayFeature.end(), [&](auto& _n)
		{

			_n = (*data)[i][numFeature];
			i++;
			return;
		});
	/*for (size_t i = 0; i < size; i++)
	{
		arrayFeature.push_back((*data)[i][numFeature]);
	}*/

	if (isMaxOrMin)
	{


		double maxValue = arrayFeature[max_element(arrayFeature.begin(), arrayFeature.end()) - arrayFeature.begin()];

		/*
		double maxValue = DBL_MIN;
		for (size_t i = 0; i < featureSize; i++)
		{
			if ((arrayFeature[i] > maxValue))
			{
				maxValue = arrayFeature[i];
			}
		}
		*/
		return maxValue;
	}
	//� ���� ������ ����������� �������
	double minValue = arrayFeature[min_element(arrayFeature.begin(), arrayFeature.end()) - arrayFeature.begin()];
	/*
	double minValue = DBL_MAX;

	for (size_t i = 0; i < featureSize; i++)
	{

		if ((*data)[i][numFeature] < minValue)
		{
			minValue = arrayFeature[i];
		}
	}
	*/
	return minValue;

}
KohonenMap::~KohonenMap()
{
	data->clear();
	weights->clear();
	delete data;
	delete weights;
	size = 0;
	featureSize = 0;
	clusterization.clear();
}


vector<size_t> KohonenMap::findWinner(size_t k)
{
	size_t i = 0;

	for_each(weights->begin(), weights->end(), [&](auto& _n)
		{
			size_t j = 0;
			double temporaryDist = accumulate(_n.begin(), _n.end(), 0.0, [&](double acc, auto& _m) //��������� ������: ������ ���������� ��� �����, ����������� �����
				{
					j++;
					return acc + pow(_m - (*data)[k][j - 1], 2);
				});
			distanceSquared[i] = temporaryDist;
			//pow(temporaryDist, 0.5); //������ ���� ���, �� ������������� �������� �� ����� ����������, ��� ��� � ������� ���������� �����
			i++;
		});


	/*for (size_t i = 0; i < numPoints; i++)  //����������� �� ���� �����
	{
		double temporaryDist = 0.0;
		for (size_t j = 0; j < featureSize; j++)  //����������� �� ���� ���������
		{
			temporaryDist += pow((*weights)[i][j] - (*data)[k][j], 2);

		}
		distanceSquared[i] = pow(temporaryDist, 0.5);
	}*/

	//sort(distanceSquared.begin(), distanceSquared.end());
	vector<double> check = { distanceSquared };
	std::sort(check.begin(), check.end());
	vector<size_t> result;

	for (size_t i = 0; i < numPoints; i++)
	{
		double winnerValue = check[i];

		auto winnerIndex = find(distanceSquared.begin(), distanceSquared.end(), winnerValue);

		const auto pos = distance(distanceSquared.begin(), winnerIndex);

		result.push_back(pos);
	}
	clusterization[result[0]] += 1;
	return result; //������ ������� �����, � �� ��������� ������
}
/*
void KohonenMap::training(int E_ALGO, int numEpoch)
{
	cout << "\n" << E_ALGO;

	size_t winnerIndex(0);

	for (size_t epoch = 1; epoch <= numEpoch; epoch++)
	{
		for (size_t i = 0; i < size; i++) //�������� �� ������� ������
		{
			winnerIndex = findWinner(i); //�������, ����� ������ ����� ����� ����� � i-�� ������
			updateWeights(i, winnerIndex, epoch);
		}
	}

	for (size_t i = 0; i < numPoints; i++)
	{
		for (size_t j = 0; j < featureSize; j++) {
			if ((*weights)[i][j] < 1e-7)
			{
				(*weights)[i][j] = 0.0;
			}
		}
	}

	return;
}
*/
void KohonenMap::countErr(vector<size_t> winnerIndex, double* err, int algorithm)
{

	if (algorithm == 0) //����� ���
	{
		double summErr = accumulate(data->begin(), data->end(), 0.0, [&](double accc, auto const& _n)
			{
				size_t j = 0;
				double temporaryErr = accumulate(_n.begin(), _n.end(), 0.0, [&](double acc, auto const& _m) //��������� ������: ������ ���������� ��� �����, ����������� �����
					{
						j++;
						return acc + pow((*weights)[winnerIndex[0]][j - 1] - _n[j - 1], 2);

					});
				return accc + temporaryErr;
			});
		summErr = 0;

		for (size_t i = 0; i < size; i++)
		{
			double temporaryError = 0.0;
			for (size_t j = 0; j < featureSize; j++)
			{
				temporaryError += pow((*weights)[winnerIndex[0]][j] - (*data)[i][j], 2);
			}
			summErr += temporaryError;
			//pow(temporaryError, 0.5); //������ ���� ���, �� ���� �������� ������� ������, ��������� ������ �������� ������� �������, ���
		}

		*err += summErr / size;
		return;
	}

}
void KohonenMap::training(int E_ALGO, const double precision)
{
#ifdef DEBUG
	precision = 20.0;
#endif
	double* err = new double(0);
	double prevErr = DBL_MAX;
	int epochAnalogue = 1; //���� �� ������������ updateWeights ������ ���
	clock_t timeReq = clock();
	ofstream out("log.txt");
	while ((abs(prevErr - *err) > precision) && (epochAnalogue < 400))
	{
		//clusterization.clear(); //����� ������� ��� ������ �� ��������� �����
		prevErr = *err;
		*err = 0;

		out << "%%%%%%%%%%%%%%%%%%%%%" << endl;
		out << "on epo = " << epochAnalogue << endl;
		for_each(weights->begin(), weights->end(), [&](auto const& _n)
			{
				out << _n[0] << endl;
			});

		/*for (auto x : (*weights))
		{
			out << x[0] << endl;

		}*/
		out << "%%%%%%%%%%%%%%%%%%%%%" << endl;
		for_each(data->begin(), data->end(), [&](auto& _n) //�������� �� ������� ������
			{
				size_t i = distance(data->begin(), find(data->begin(), data->end(), _n));
				vector<size_t> winnerIndex = { (*this).findWinner(i) };  //�������, ����� ������ ����� ����� ����� � i-�� ������

				(*this).updateWeights(i, winnerIndex, epochAnalogue);
				(*this).countErr(winnerIndex, err, 0); //��������� ������: ��� 0 - �� ���
			});

		/*for (size_t i = 0; i < size; i++) //�������� �� ������� ������
		{
			vector<size_t> winnerIndex = { findWinner(i) };  //�������, ����� ������ ����� ����� ����� � i-�� ������

			(*this).updateWeights(i, winnerIndex, epochAnalogue);
			(*this).countErr(winnerIndex, err, 0); //��������� ������: ��� 0 - �� ���
		}*/
		epochAnalogue++;
		if (! (epochAnalogue % 50))
		{
			timeReq = clock() - timeReq;
			cout << "It took " << timeReq << "to train our network for " << epochAnalogue << " epochs" << endl;
			cout << "Overall, the error is: " << *err << endl;
		}
		cout << prevErr - *err << endl;
#ifdef DEBUG
		break;
#endif

	}
	delete err;
	for_each(weights->begin(), weights->end(), [&](auto& _n)
		{
			auto g = bind(less<double>(), _1, 1e-7); //���������� � 1e-7
			replace_if(_n.begin(), _n.end(), g, 0.0); //���� ������������, �������� �� 0
			return;
		});

	/*for (size_t i = 0; i < numPoints; i++)
	{
		for (size_t j = 0; j < featureSize; j++) {
			if ((*weights)[i][j] < 1e-7)
			{
				(*weights)[i][j] = 0.0;
			}
		}
	}
	*/
	out.close();
	isTeached = true;
	return;
}

void KohonenMap::updateWeights(size_t i, vector<size_t> winnerIndex, int epoch)
{
	double coeff = 1.0;
	if (epoch > 1) { coeff = 6.0; }

	for (size_t j = 0; j < featureSize; j++) //���� ����� �������� ����� for
	{
		for (size_t k = 0; k < winnerIndex.size(); k++)
		{
			(*weights)[winnerIndex[k]][j] += (1 / pow(2, k)) * neighbourFunc(winnerIndex[k], j, epoch, 2) * ((*data)[i][j] - (*weights)[winnerIndex[k]][j]);
			if ((*weights)[winnerIndex[k]][j] > 1.0)
			{
				(*weights)[winnerIndex[k]][j] = 1.0;
			}
		}
	}
	return;
}
double KohonenMap::alpha(int epoch)
{
	double res = 100.0 + epoch;
	return 100.0 / res;
}

double KohonenMap::neighbourFunc(int winnerIndex, int featureIndex, int epoch, double sigma)
{
	/*double normSquared = accumulate(weights->begin(), weights->end(), 0.0, [&](double _m, auto& _n)
		{

			return _m + pow((*weights)[winnerIndex][featureIndex] - _n[featureIndex], 2);
		});
		*/


		/*for (size_t i = 0; i < numPoints; i++)
		{
			normSquared += pow((*weights)[winnerIndex][featureIndex] - (*weights)[i][featureIndex], 2);
		}*/
		//cout << alpha(epoch) * exp(-normSquared / (2 * pow(10, 2))) << endl;

	return alpha(epoch);
	//* exp(-normSquared / (2 * pow(sigma, 2)));


}

void const KohonenMap::printClusterization()
{
	cout << "********************************" << endl;
	for_each(clusterization.begin(), clusterization.end(), [](auto const& _n)
		{
			cout << "unit number " << _n.first << " had took " << _n.second << " samples in it" << endl;
			cout << "********************************" << endl;
		});
	/*for (auto it = clusterization.begin(); it != clusterization.end(); ++it)
	{
		cout << "unit number " << it->first << " had took " << it->second << " samples in it" << endl;
		cout << "********************************" << endl;
	}*/
}

void const KohonenMap::print(int featureIndex)
{
	cout << endl;
	for_each(weights->begin(), weights->end(), [&](auto const& _n)
		{
			cout << _n[featureIndex] << endl;
			return;
		});
	/*
	for (size_t i = 0; i < numPoints; i++)
	{
		cout <<  (*weights)[i][featureIndex] << endl;
	}
	*/
	return;
}

void KohonenMap::writeToFile()
{

	ofstream out("KohonenMap.txt");
	if (out.is_open())
	{
		for_each(weights->begin(), weights->end(), [&](auto const& _n)
			{
				size_t i = distance(weights->begin(), find(weights->begin(), weights->end(), _n));
				vector<size_t> winnerIndex = { (*this).findWinner(i) };  //�������, ����� ������ ����� ����� ����� � i-�� ������

				for_each(_n.begin(), _n.end(), [&](auto const& _m)
					{
						out << _m << " ";

					});
				for (size_t k = 0; k < howMuchInOneCell; k++)
				{
					out << namesData[winnerIndex[k]] << " ,"; //����� �� �������� ����� �������
				}
				out << "\n";
			});

		/*for (size_t i = 0; i < numPoints; i++)
		{
			for (size_t j = 0; j < featureSize; j++) {
				out << (*weights)[i][j] << " ";
			}
			out << "\n";

		}*/
		cout << "Program ended successefully!\n";
	}
	else
	{
		cout << "Error while reading file!\n";
	}
	out.close();
}

KohonenMap::KohonenMap(string fileToReadWeights) : isTeached(true)
{

	data = new vector<vector<double>>;
	weights = new vector<vector<double>>;
	notChangedSamplesNames = new vector<string>(100, "");
	if (fileToReadWeights != "")
	{
		ifstream file1(fileToReadWeights);
		char delim = ' ';
		char delim1 = ',';
		size_t k = 0;
		if (file1.is_open())
		{
			string line, feature, name = "";

			while (getline(file1, line))
			{
				if (line == "") { break; }

				namesCells.push_back({});
				vector<double> weightsRow;
				stringstream sline(line);

				while (getline(sline, feature, ' '))
				{

					if (feature[0] == ',')
					{
						namesCells[k].push_back(name);

						string afterComma;
						string toBeSkipped;
						getline(sline, afterComma, delim);
						toBeSkipped.insert(toBeSkipped.begin(), feature.begin() + 1, feature.end());
						toBeSkipped.push_back('_');
						toBeSkipped.insert(toBeSkipped.end(), afterComma.begin(), afterComma.end());
						feature.clear();
						copy(toBeSkipped.begin(), toBeSkipped.end(), back_inserter(feature));

						name.clear();
					}

					if (check(feature[0]))
					{
						weightsRow.push_back(stod(feature));
					}
					else
					{
						if (name != "")
						{
							name += '_';
						}
						name += feature;
					}
				}

				weights->push_back(weightsRow);
				k++;
				name.clear();
			}

			size = weights->size();
			numPoints = size;
			featureSize = (*weights)[0].size();

		}
		else
		{
			cerr << "Error while opening file with weights to read!\n";
			exit(1);
		}
	}


	fill_n(back_inserter(distanceSquared), numPoints, 0.0);
}

void KohonenMap::readData(string fromWhere, bool _countHonkong)
{
	Normalizer A(fromWhere);
	if (_countHonkong)
	{
		A.convertHonkongToChina();

	}
	A.normalize(data);
	namesData = { A.getNames() };
	random_device rd;
	mt19937 g(rd());
	size_t dataSize = data->size();
	vector<size_t> index(dataSize);
	iota(index.begin(), index.end(), 0); //������ ������ ��������, ������� � 0
	shuffle(index.begin(), index.end(), g); //������ ��� ���������

	vector<vector<double>>* dataSafe = new vector<vector<double>>(dataSize, vector<double>(featureSize)); //��������� ������������� �� ������ � ���� �� ���������
	vector<string> namesSafe = vector<string>(dataSize);
	for_each(index.begin(), index.end(), [&](auto& _n)
		{
			size_t i = distance(index.begin(), find(index.begin(), index.end(), _n));

			(*dataSafe)[i] = (*data)[_n]; //��� �����, ����������, �������������� �� ������ ������
			namesSafe[i] = namesData[_n];
		});

	copy(dataSafe->begin(), dataSafe->end(), data->begin()); //������ ������� ���� � ���
	copy(namesSafe.begin(), namesSafe.end(), namesData.begin());

	howMuchInOneCell = dataSize / size; //��, ������� ������� ����� � ������ ������
	return;
}
void KohonenMap::clusterize(const size_t whichOrder, const double percentOfChange, bool isRanJustAfterTrain)
{
	try
	{
		if (!isTeached) { throw "You forgot to train the network!\n"; }
	}
	catch (string e)
	{
		cerr << "Error: " << e;
		exit(1);
	}





	if (!isRanJustAfterTrain) //���� ��� ������� �������� ����� ����� �������� � ���� ������ �������� ����� �� ������, ������� �� �������������
	{

		/*try
		{
			if (size != numPoints)
			{
				throw 1.0; //��� ������ ����������������, ���� ����� �������
			}
			data->clear();
			copy(weights->begin(), weights->end(), data->end()); //�������� ����������� ���� � ����, ����� �� ������������ ������� ������
		}
		catch (double er1)
		{
			cerr << "Can't transform data into weights for classification: try check dimension";
			exit(1);
		}
		catch (const bad_alloc& e)
		{
			cerr << "Can't transform data into weights for classification: " << e.what();
			exit(1);

		}
		catch (const exception& e)
		{
			cerr << "Can't transform data into weights for classification:  " << e.what();
			exit(1);
		}

		weights->clear(); //� ������ ����� ������ ����� �������� �����
		*/
	}
	copy(namesData.begin() + (whichOrder - 1) * 100, namesData.begin() + (whichOrder) * 100, notChangedSamplesNames->begin());
	for_each(notChangedSamplesNames->begin(), notChangedSamplesNames->end(), [&](auto& _n)
		{
			replace(_n.begin(), _n.end(), ' ', '_');
		});



	vector<vector<double>> testSet = { littleChange(whichOrder, percentOfChange) }; //������ �������� ����� �� whichOrder'��� ����� ������� ��������� ������� 
	//� � ����� ������� ��������� ���� ����� ������ �������� ��������

	try
	{
		data->clear();
		namesData.clear();
		size_t i = 0;
		for_each(testSet.begin(), testSet.end(), [&](auto& _n)
			{
				data->push_back({});
				i++;
				(*data)[i - 1].insert((*data)[i - 1].end(), _n.begin(), _n.end());
			});
		testSet.clear();
	}
	catch (const exception& e)
	{
		cerr << "Can't transform weights into test set for classification:  " << e.what();
		exit(1);
	}

	//���� �� ����������
	size_t i = 0;
	for_each(data->begin(), data->end(), [&](auto const& _n)
		{
			i++;

			size_t cellBest = (*this).findWinner(i - 1)[0]; //����� ������ �������������� ���� �����
			//vecBest.erase(vecBest.begin() + howMuchInOneCell, vecBest.end()); //��� ������ ���� ������� ���, �� ������� ������ �������������

			resultOfWork[i - 1] = { namesCells[cellBest] }; //��������� ��� � ������� �������
			weights->erase(weights->begin() + cellBest); //������� ��������� �� �������
			size--;
			numPoints--;
			namesCells.erase(namesCells.begin() + cellBest);
			distanceSquared.erase(distanceSquared.begin() + cellBest);

		});

	string nameFile1 = to_string(whichOrder * 100);
	string nameFile2 = to_string((whichOrder - 1) * 100);
	string name = "KohonenClassifier" + nameFile2 + "_" + nameFile1 + ".txt";
	ofstream out2(name);
	if (out2.is_open())
	{
		for_each(resultOfWork.begin(), resultOfWork.end(), [&](auto const& _n)
			{
				out2 << _n.first << " ";
				for_each(_n.second.begin(), _n.second.end(), [&](auto const& _m)
					{
						out2 << _m << " ";
					});
				out2 << endl;
			});
		out2 << precisionRate();
	}
	else
	{
		cerr << "Error while writing test set output!\n";
	}
	out2.close();
	notChangedSamplesNames->clear();
	delete notChangedSamplesNames;
	return;
}
bool KohonenMap::check(char symb) const
{
	if (((static_cast<int>(symb) > 47) && (static_cast<int>(symb) < 58))
		|| (symb == ' ') || (symb == '\0'))
	{
		return true;
	}
	return false;
}
double KohonenMap::precisionRate()
{
	double rate = 0;
	vector<vector<string>> res(100, vector<string>(howMuchInOneCell));
	auto x = resultOfWork.begin();
	for (size_t i = 0; i < res.size(); i++)
	{
		for (size_t k = 0; k < howMuchInOneCell; k++)
		{
			res[i][k] = x->second[k];
		}

		x = next(resultOfWork.begin(), i + 1);
	}

	size_t i = 0;
	for_each(notChangedSamplesNames->begin(), notChangedSamplesNames->end(), [&](auto& _n)
		{
			i++;//���� ����� ������������� ��� �������� (��������������� �������� � ��������� �� ������, � ��� �� howMuchInOneCell ���

			rate += count(res[i - 1].begin(), res[i - 1].end(), _n);

		});

	rate /= 100;
	return rate;

}

/*
__global__ void AddIntsCUDA(int* a, int* b)
{
	a[0] += b[0];
}
*/
/*
int main() {
	int a = 5, b = 9;
	int* d_a, *d_b;

	hipMalloc(&d_a, sizeof(int));
	hipMalloc(&d_b, sizeof(int));

	hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

	AddIntsCUDA<<<1, 1>>>(d_a, d_b);

	hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&b, d_b, sizeof(int), hipMemcpyDeviceToHost);

	cout << a << endl;
	hipFree(d_a);
	hipFree(d_b);
	return 0;
}
*/
